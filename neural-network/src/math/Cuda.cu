#include "hip/hip_runtime.h"
#include "_Math.h"

#include "../Base.h"

namespace Utils {

        __device__ inline float min(float a, float b)
        {
                return a < b ? a : b;
        }

        __device__ inline float max(float a, float b)
        {
                return a > b ? a : b;
        }

        __device__ inline float tanh(float x)
        {
                if (fabsf(x) >= 4.9f)
                        return copysignf(1.0f, x);

                const float x2 = x * x;
                return x * (135135.0f + x2 * (17325.0f + x2 * (378.0f + x2))) /
                       (135135.0f + x2 * (62370.0f + x2 * (3150.0f + x2 * 28.0f)));
        }

        __device__ inline float tanhDerivative(float x)
        {
                if (x == 0.0f)
                        return 1.0f;

                if (fabsf(x) > 4.9f)
                        return 0.0f;

                const float tanh = Utils::tanh(x);
                return 1.0f - tanh * tanh;
        }

}

namespace Kernels {

        __global__ void sum(uint32_t size, const float first[], const float second[], float result[])
        {
                const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

                if (idx < size)
                        result[idx] = first[idx] + second[idx];
        }

        __global__ void sub(uint32_t size, const float first[], const float second[], float result[])
        {
                const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

                if (idx < size)
                        result[idx] = first[idx] - second[idx];
        }

        __global__ void mul(uint32_t size, const float first[], const float second[], float result[])
        {
                const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
                
                if (idx < size)
                        result[idx] = first[idx] * second[idx];
        }

        __global__ void div(uint32_t size, const float first[], const float second[], float result[])
        {
                const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
                
                if (idx < size)
                        result[idx] = first[idx] / second[idx];
        }

        __global__ void sum(uint32_t size, const float first[], float scalar, float result[])
        {
                const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
                
                if (idx < size)
                        result[idx] = first[idx] + scalar;
        }

        __global__ void sub(uint32_t size, const float first[], float scalar, float result[])
        {
                const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
                
                if (idx < size)
                        result[idx] = first[idx] - scalar;
        }

        __global__ void mul(uint32_t size, const float first[], float scalar, float result[])
        {
                const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
                
                if (idx < size)
                        result[idx] = first[idx] * scalar;
        }

        __global__ void div(uint32_t size, const float first[], float scalar, float result[])
        {
                const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
                
                if (idx < size)
                        result[idx] = first[idx] / scalar;
        }

        __global__ void tanh(uint32_t size, const float data[], float result[])
        {
                const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
                
                if (idx < size)
                        result[idx] = Utils::tanh(data[idx]);
        }

        __global__ void tanh_derivative(uint32_t size, const float data[], float result[])
        {
                const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
                
                if (idx < size)
                        result[idx] = Utils::tanhDerivative(data[idx]);
        }

        __global__ void ReLU(uint32_t size, const float data[], float result[])
        {
                const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
                
                if (idx < size)
                        result[idx] = Utils::max(0.0f, data[idx]);
        }

        __global__ void ReLU_derivative(uint32_t size, const float data[], float result[])
        {
                const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
                
                if (idx < size)
                        result[idx] = data[idx] >= 0.0f ? 1.0f : 0.0f;
        }

        __global__ void min(uint32_t size, const float a[], float min, float result[])
        {
                const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
                
                if (idx < size)
                        result[idx] = Utils::min(a[idx], min);
        }

        __global__ void max(uint32_t size, const float a[], float max, float result[])
        {
                const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
                
                if (idx < size)
                        result[idx] = Utils::max(a[idx], max);
        }

        __global__ void clamp(uint32_t size, const float data[], float min, float max, float result[])
        {
                const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
                
                if (idx < size)
                        result[idx] = Utils::min(Utils::max(data[idx], min), max);
        }

        __global__ void min(uint32_t size, const float a[], const float b[], float result[])
        {
                const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
                
                if (idx < size)
                        result[idx] = Utils::min(a[idx], b[idx]);
        }

        __global__ void max(uint32_t size, const float a[], const float b[], float result[])
        {
                const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
                
                if (idx < size)
                        result[idx] = Utils::max(a[idx], b[idx]);
        }

        __global__ void clamp(uint32_t size, const float data[], const float min[], const float max[], float result[])
        {
                const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
                
                if (idx < size)
                        result[idx] = Utils::min(Utils::max(data[idx], min[idx]), max[idx]);
        }

        __global__ void matvec_mul(uint32_t width, uint32_t height, const float matrix[], const float vector[], float result[])
        {
                const uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
                if (row >= height)
                        return;

                float sum = 0.0f;
                for (uint32_t col = 0; col < width; col++)
                        sum += matrix[row * width + col] * vector[col];

                result[row] = sum;
        }

} // namespace Kernels

#define DECLARE_CUDA_FUNCTION(__name__, __size__, ...)                                                          \
template<> void _Math<MATH_CUDA>:: __name__ (GET_ARGS(__VA_ARGS__))                                             \
{                                                                                                               \
        const uint32_t BLOCKS_COUNT = ((__size__) + BLOCK_SIZE - 1) >> BLOCK_BITSHIFT;                          \
        Kernels:: __name__ <<<BLOCKS_COUNT, BLOCK_SIZE>>>(GET_ARGS_NAMES(__VA_ARGS__));                         \
        CUDA_CHECK_ERROR(hipGetLastError(), "Kernels::" #__name__ " launch failed.");                          \
        CUDA_CHECK_ERROR(hipDeviceSynchronize(), "Error synchronizing in _Math<MATH_CUDA>::" #__name__);       \
}

DECLARE_CUDA_FUNCTION(sum, size, uint32_t, size, const float *, first, const float *, second, float *, result)
DECLARE_CUDA_FUNCTION(sub, size, uint32_t, size, const float *, first, const float *, second, float *, result)
DECLARE_CUDA_FUNCTION(mul, size, uint32_t, size, const float *, first, const float *, second, float *, result)
DECLARE_CUDA_FUNCTION(div, size, uint32_t, size, const float *, first, const float *, second, float *, result)
DECLARE_CUDA_FUNCTION(sum, size, uint32_t, size, const float *, first, float, scalar, float *, result)
DECLARE_CUDA_FUNCTION(sub, size, uint32_t, size, const float *, first, float, scalar, float *, result)
DECLARE_CUDA_FUNCTION(mul, size, uint32_t, size, const float *, first, float, scalar, float *, result)
DECLARE_CUDA_FUNCTION(div, size, uint32_t, size, const float *, first, float, scalar, float *, result)
DECLARE_CUDA_FUNCTION(tanh, size, uint32_t, size, const float *, data, float *, result)
DECLARE_CUDA_FUNCTION(tanh_derivative, size, uint32_t, size, const float *, data, float *, result)
DECLARE_CUDA_FUNCTION(ReLU, size, uint32_t, size, const float *, data, float *, result)
DECLARE_CUDA_FUNCTION(ReLU_derivative, size, uint32_t, size, const float *, data, float *, result)
DECLARE_CUDA_FUNCTION(min, size, uint32_t, size, const float *, first, const float *, second, float *, result)
DECLARE_CUDA_FUNCTION(max, size, uint32_t, size, const float *, first, const float *, second, float *, result)
DECLARE_CUDA_FUNCTION(clamp, size, uint32_t, size, const float *, data, const float *, min, const float *, max, float *, result)
DECLARE_CUDA_FUNCTION(min, size, uint32_t, size, const float *, data, float, min, float *, result)
DECLARE_CUDA_FUNCTION(max, size, uint32_t, size, const float *, data, float, max, float *, result)
DECLARE_CUDA_FUNCTION(clamp, size, uint32_t, size, const float *, data, float, min, float, max, float *, result)
DECLARE_CUDA_FUNCTION(matvec_mul, width, uint32_t, width, uint32_t, height, const float *, matrix, const float *, vector, float *, result)